
#include <hip/hip_runtime.h>
#include <stdio.h>
#define kNumIterations 10

// kernel launch overheads

__global__ void emptyKernel()
{
}

void test_time_emptykernel_overhead()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    for (int i = 0; i < kNumIterations; i++)
    {
        emptyKernel<<<1, 1>>>();
        hipDeviceSynchronize();
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    milliseconds /= kNumIterations;
    printf("Time to launch an empty kernel: %f ms\n", milliseconds);
}

// memory allocation overheads



// memory copy overheads

void run_tests()
{
    // kernel launch overheads
    test_time_emptykernel_overhead();
}