
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void simpleKernel()
{
    printf("Hello, World!\n");
}

int caller()
{
    simpleKernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}