#include <stdio.h>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>

#include <exception>
#define kNumIterations 10

#define TIMER_EVENTS_CREATE \
    hipEvent_t start, stop; \
    checkCudaErrors(hipEventCreate(&start)); \
    checkCudaErrors(hipEventCreate(&stop));

#define TIMER_EVENTS_RECORD_START \
    checkCudaErrors(hipEventRecord(start));

#define TIMER_EVENTS_RECORD_STOP \
    checkCudaErrors(hipEventRecord(stop)); \
    checkCudaErrors(hipEventSynchronize(stop));

#define TIMER_EVENTS_DESTROY \
    checkCudaErrors(hipEventDestroy(start)); \
    checkCudaErrors(hipEventDestroy(stop));

#define REPORT_TIME(label) \
    float milliseconds = 0; \
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop)); \
    milliseconds /= kNumIterations; \
    printf("Time to %s: %f ms\n", label, milliseconds);

#define checkCudaErrors(ARG)                                                \
    if (hipError_t const err = ARG; err != hipSuccess)                     \
    {                                                                        \
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));        \
        exit(EXIT_FAILURE);                                                  \
    }

constexpr int kOneM = 1024 * 1024;
constexpr int kHundredM = 1024 * 1024 * 100;
constexpr int kOneG = 1024 * 1024 * 1024;

// kernel launch overheads

__global__ void emptyKernel()
{
}

void TIME_emptykernel_overhead()
{
    hipEventCreateEATE
    TIMER_EVENTS_RECORD_START
    
    for (int i = 0; i < kNumIterations; i++)
    {
        emptyKernel<<<1, 1>>>();
        hipDeviceSynchronize();
    }

    TIMER_EVENTS_RECORD_STOP

    REPORT_TIME("launch an empty kernel")
}

// memory allocation overheads
void time_allocation_overhead(int num_bytes, std::string label, int numIterations = kNumIterations)
{
    std::vector<char*> d_dataVec(numIterations, nullptr);

    hipEventCreateEATE
    TIMER_EVENTS_RECORD_START

    for (int i = 0; i < numIterations; i++)
        checkCudaErrors(hipMalloc(&d_dataVec[i], num_bytes));

    TIMER_EVENTS_RECORD_STOP
    REPORT_TIME(label.c_str())
    
    for (int i = 0; i < numIterations; i++)
    checkCudaErrors(hipFree(d_dataVec[i]));
}

#define TIME_ALLOCATION_OVERHEAD(SIZE, DESC) \
    void TIME_##SIZE##byte_allocation_overhead()  \
    {                                         \
        time_allocation_overhead(SIZE, DESC); \
    }

TIME_ALLOCATION_OVERHEAD(1, "allocate 1 byte")
TIME_ALLOCATION_OVERHEAD(1024, "allocate 1 KB")
TIME_ALLOCATION_OVERHEAD(kOneM, "allocate 1 MB")
TIME_ALLOCATION_OVERHEAD(kHundredM, "allocate 100 MB")
TIME_ALLOCATION_OVERHEAD(kOneG, "allocate 1 GB")

// memory copy overheads

void time_copy_overhead(int num_bytes, std::string label, int numIterations = kNumIterations)
{
    char *h_data = new char[num_bytes], *d_data = nullptr;
    checkCudaErrors(hipMalloc(&d_data, num_bytes));

    hipEventCreateEATE
    TIMER_EVENTS_RECORD_START

    for (int i = 0; i < numIterations; i++)
    {
        checkCudaErrors(hipMemcpy(d_data, h_data, num_bytes, hipMemcpyHostToDevice));
    }

    TIMER_EVENTS_RECORD_STOP
    REPORT_TIME(label.c_str())

    delete[] h_data;
    checkCudaErrors(hipFree(d_data));
}

#define TIME_COPY_OVERHEAD(SIZE, DESC) \
    void TIME_##SIZE##byte_copy_overhead()  \
    {                                         \
        time_copy_overhead(SIZE, DESC); \
    }

TIME_COPY_OVERHEAD(1, "copy 1 byte")
TIME_COPY_OVERHEAD(1024, "copy 1 KB")
TIME_COPY_OVERHEAD(kOneM, "copy 1 MB")
TIME_COPY_OVERHEAD(kHundredM, "copy 100 MB")
TIME_COPY_OVERHEAD(kOneG, "copy 1 GB")

// main test runner

void run_tests()
{
    // kernel launch overheads
    TIME_emptykernel_overhead();

    // memory allocation overheads
    TIME_1byte_allocation_overhead();
    TIME_1024byte_allocation_overhead();
    TIME_kOneMbyte_allocation_overhead();
    TIME_kHundredMbyte_allocation_overhead();
    TIME_kOneGbyte_allocation_overhead();

    // memory copy overheads
    TIME_1byte_copy_overhead();
    TIME_1024byte_copy_overhead();
    TIME_kOneMbyte_copy_overhead();
    TIME_kHundredMbyte_copy_overhead();
    TIME_kOneGbyte_copy_overhead();
}